
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define SIZE 128
#include <sys/time.h>

__global__ void scan(int* input, int* output){

  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
 // int lindex = threadIdx.x;

  // accounting for thread size being less than array size  
  if (gindex >= SIZE){
  	return;
  }

  int* source;
  int* destination; 

  source = &input[0];
  destination = &output[0];
  int* temp;

  // j is my stride, threadIdx is gindex
  for (int j=1; j <= SIZE; j*=2) {
    __syncthreads();
    if (gindex < j){
     	destination[gindex] = source[gindex];
     }
     else{
	    destination[gindex] = source[gindex] + source[gindex-j];
		temp = destination;
		destination = source;
		source = temp;
	}
  }
  output[gindex] = source[gindex];
 
  //__syncthreads();
}



double get_clock() {
  struct timeval tv; 
  int ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { 
  	printf("gettimeofday error"); 
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void) {
  // int N = 100;
  // Our SIZE is N (array size)
 
  int *input, *output;
  
  double t0 = get_clock();

  
 // allocate memory
	hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));
  
  // initialize inputs
   for (int i = 0; i < SIZE; i++) {
    input[i] = 1;
   }
	
  // run the kernel
  scan<<<1,128>>>(input, output); 

  // synchronize 
  hipDeviceSynchronize();

  double t1 = get_clock();
  printf("time per call: %f ns\n", t1-t0);

   // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");


  printf("%s\n", hipGetErrorString(hipGetLastError()));
	
  // free mem
  hipFree(input);
  hipFree(output);

  return 0;
}
