
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define SIZE 128
#include <sys/time.h>

__global__ void scan(int* input, int* output){

  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
 // int lindex = threadIdx.x;

  // accounting for thread size being less than array size  
  if (gindex >= SIZE){
  	return;
  }

  // do the scan
  for (int i = 0;  i < SIZE; i++) {
   int value = 0;
   for (int j = 0; j <= i; j++) {
     value += input[j];
   }
    output[i] = value;
  }	

 
  __syncthreads();
}



double get_clock() {
  struct timeval tv; 
  int ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { 
  	printf("gettimeofday error"); 
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void) {
  // int N = 100;
  // Our SIZE is N (array size)
 
  int *input, *output;
  
  double t0 = get_clock();

  
 // allocate memory
	hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));
  
  // initialize inputs
   for (int i = 0; i < SIZE; i++) {
    input[i] = 1;
   }
	
  // run the kernel
  scan<<<1,128>>>(input, output); 

  // synchronize 
  hipDeviceSynchronize();

  double t1 = get_clock();
  printf("time per call: %f ns\n", t1-t0);

   // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");


  printf("%s\n", hipGetErrorString(hipGetLastError()));
	
  // free mem
  hipFree(input);
  hipFree(output);

  return 0;
}