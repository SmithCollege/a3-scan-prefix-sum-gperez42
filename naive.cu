
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define SIZE 100
#define BLOCK_SIZE 128
#include <sys/time.h>

__global__ void scan(int* input, int* output){

  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
 // int lindex = threadIdx.x;

  // accounting for thread size being less than array size  
  if (gindex >= SIZE){
  	return;
  }

  // do the scan
  for (int i = 0;  i < SIZE; i++) {
   int value = 0;
   for (int j = 0; j <= i; j++) {
     value += input[j];
   }
    output[i] = value;
  }	

 
  __syncthreads();
}



double get_clock() {
  struct timeval tv; 
  int ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { 
  	printf("gettimeofday error"); 
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void) {
  // int N = 100;
  // Our SIZE is N (array size)
 
  int *input, *output;
  int x; // Number of blocks to launch

  
  double t0 = get_clock();

  
 // allocate memory
	hipMallocManaged(&input, SIZE*sizeof(int));
    hipMallocManaged(&output, SIZE*sizeof(int));
  
  // initialize inputs
   for (int i = 0; i < SIZE; i++) {
    input[i] = 1;
   }
	
  // run the kernel
  // scan<<<1,128>>>(input, output); 
  
    // Check if SIZE is a multiple of BLOCK_SIZE
    if (SIZE % BLOCK_SIZE != 0) {
      // If not a perfect multiple, calculate the number of blocks needed
      if (SIZE > BLOCK_SIZE) {
        x = SIZE / BLOCK_SIZE + 1; // Add 1 if not perfectly divisible
        printf("Number of blocks (with extra): %d\n", x);
      }
    } else {
      x = SIZE / BLOCK_SIZE; // Perfectly divisible case
      printf("Number of blocks (perfectly divisible): %d\n", x);
    }
  // Launch the kernel with the calculated number of blocks
    scan<<<x, BLOCK_SIZE>>>(input, output);

  // synchronize 
  hipDeviceSynchronize();

  double t1 = get_clock();
  printf("time per call: %f ns\n", t1-t0);

   // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");


  printf("%s\n", hipGetErrorString(hipGetLastError()));
	
  // free mem
  hipFree(input);
  hipFree(output);

  return 0;
}
