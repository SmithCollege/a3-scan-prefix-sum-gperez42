
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define SIZE 128
#include <sys/time.h>

__global__ void scan(int* input, int* output){

  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
 // int lindex = threadIdx.x;

  // accounting for thread size being less than array size  
  if (gindex >= SIZE){
  	return;
  }

  // do the scan
  for (int i = SIZE;  i < SIZE; i++) {
   int value = 0;
   for (int j = 0; j <= i; j++) {
     value += input[j];
   }
    output[i] = value;
  }	

 
  __syncthreads();
}



double get_clock() {
  struct timeval tv; 
  int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { 
  	printf("gettimeofday error"); 
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int N;
//int* times;

int main(void) {
  // int N = 100;
  // int i;
  // int *input, *output
  
  double t0 = get_clock();

  /*
 // allocate memory
	cudaMallocManaged(&input, N*sizeof(int));
    cudaMallocManaged(&output, N*sizeof(int));
 */
 
  // allocate memory
  int* input = (int*) malloc(sizeof(int) * SIZE);
  int* output = (int*) malloc(sizeof(int) * SIZE);

  
  // initialize inputs
   for (int i = 0; i < SIZE; i++) {
    input[i] = 1;
   }
	
  // run the kernel
  scan<<<1, 128>>>(input, output); 

  // synchronize 
  hipDeviceSynchronize();

  double t1 = get_clock();
  printf("time per call: %f ns\n", t1-t0);

   // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");

  // free mem
  hipFree(input);
  hipFree(output);

  return 0;
}
