
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 128
#include <sys/time.h>

double get_clock() {
  struct timeval tv; 
  int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { 
  	printf("gettimeofday error"); 
  }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int N;
int* times;

int main() {
  int i;
  
  double t0 = get_clock();
  for (i=0; i<N; i++) {
  	times[i] = get_clock();
  }
  
  // allocate memory
  int* input = (int*) malloc(sizeof(int) * SIZE);
  int* output = (int*) malloc(sizeof(int) * SIZE);

  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    input[i] = 1;
   }

  // do the scan
  for (int i = 0; i < SIZE; i++) {
   int value = 0;
   for (int j = 0; j <= i; j++) {
     value += input[j];
   }
    output[i] = value;
  }

  // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");

  // synchronize 
  hipDeviceSynchronize();

  double t1 = get_clock();
  printf("time per call: %f ns\n", t1-t0);

  // free mem
  free(input);
  free(output);

  return 0;
}
